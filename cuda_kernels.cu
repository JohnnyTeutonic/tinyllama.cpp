#include "hip/hip_runtime.h"
/**
 * @file cuda_kernels.cu
 * @brief CUDA kernel implementations for TinyLlama GPU operations
 *
 * This file contains the actual CUDA kernel implementations for the GPU-accelerated
 * operations declared in cuda_kernels.h. The implementations are optimized for
 * NVIDIA GPUs and use CUDA-specific features for maximum performance.
 *
 * Key implementation details:
 * - Uses shared memory for efficient reduction operations
 * - Implements parallel reduction patterns for normalization
 * - Leverages cuBLAS for matrix operations
 * - Supports both FP32 and BF16 formats
 * - Includes error checking and memory management
 */

#include "cuda_kernels.h"
#include "model_macros.h"

#ifdef HAS_CUDA

#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include <cmath>
#include <iostream>

/**
 * @brief Converts BF16 to FP32 on the device
 * 
 * This device function converts Brain Floating Point (BF16) values to FP32.
 * It uses CUDA's native BF16 support on architectures >= 800, and falls back
 * to a manual conversion on older architectures.
 * 
 * @param bf16_raw Raw BF16 value as uint16_t
 * @return Converted FP32 value
 */
__device__ inline float bf16_to_float32_device(uint16_t bf16_raw) {
#if __CUDA_ARCH__ >= 800
  __hip_bfloat16 bf16_val;

  memcpy(&bf16_val, &bf16_raw, sizeof(uint16_t));
  return __bfloat162float(bf16_val);
#else

  unsigned int bits = ((unsigned int)bf16_raw) << 16;
  float result;
  memcpy(&result, &bits, sizeof(float));
  return result;
#endif
}

/**
 * @brief Kernel for computing sum of squares in RMS normalization
 * 
 * This kernel computes the sum of squares of input elements using parallel
 * reduction with shared memory. It's the first step in RMS normalization.
 * 
 * @param x Input tensor
 * @param partial_sums Output array for partial sums
 * @param n Size of input tensor
 */
__global__ void rmsnorm_sum_squares_kernel(const float* x, float* partial_sums,
                                           int n) {
  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? x[i] * x[i] : 0.0f;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    partial_sums[blockIdx.x] = sdata[0];
  }
}

/**
 * @brief Kernel for applying RMS normalization
 * 
 * This kernel applies the normalization weights and scaling factor to the
 * input tensor. It's the second step in RMS normalization.
 * 
 * @param x Input tensor
 * @param weight Normalization weights
 * @param out Output tensor
 * @param n Size of tensors
 * @param inv_norm_factor Inverse of the normalization factor
 */
__global__ void rmsnorm_apply_kernel(const float* x, const float* weight,
                                     float* out, int n, float inv_norm_factor) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    out[i] = x[i] * inv_norm_factor * weight[i];
  }
}

/**
 * @brief Implementation of RMS normalization with device pointers
 * 
 * This function implements the complete RMS normalization process:
 * 1. Computes sum of squares using parallel reduction
 * 2. Calculates normalization factor
 * 3. Applies normalization with weights
 * 
 * @param x_dev Input tensor (device pointer)
 * @param weight_dev Normalization weights (device pointer)
 * @param out_dev Output tensor (device pointer)
 * @param n Size of tensors
 * @param eps Epsilon for numerical stability
 * @param stream CUDA stream for asynchronous execution
 */
void rmsnorm_vector_cuda(const float* x_dev, const float* weight_dev,
                         float* out_dev, int n, float eps,
                         hipStream_t stream) {
  const int threads_per_block = 256;
  int num_blocks_reduce = (n + threads_per_block - 1) / threads_per_block;
  size_t shared_mem_size = threads_per_block * sizeof(float);

  float* partial_sums_dev = nullptr;
  gpuErrchk(hipMalloc(&partial_sums_dev, num_blocks_reduce * sizeof(float)));

  rmsnorm_sum_squares_kernel<<<num_blocks_reduce, threads_per_block,
                               shared_mem_size, stream>>>(x_dev,
                                                          partial_sums_dev, n);
  gpuErrchk(hipGetLastError());

  float* partial_sums_host = new float[num_blocks_reduce];
  gpuErrchk(hipMemcpy(partial_sums_host, partial_sums_dev,
                       num_blocks_reduce * sizeof(float),
                       hipMemcpyDeviceToHost));

  double total_ssq = 0.0;
  for (int i = 0; i < num_blocks_reduce; ++i) {
    total_ssq += partial_sums_host[i];
  }
  total_ssq /= n;
  float inv_norm_factor = 1.0f / SAFE_SQRT(static_cast<float>(total_ssq) + eps);
  delete[] partial_sums_host;
  gpuErrchk(hipFree(partial_sums_dev));

  int num_blocks_apply = (n + threads_per_block - 1) / threads_per_block;
  rmsnorm_apply_kernel<<<num_blocks_apply, threads_per_block, 0, stream>>>(
      x_dev, weight_dev, out_dev, n, inv_norm_factor);
  gpuErrchk(hipGetLastError());
}

void rmsnorm_vector_cuda(const std::vector<float>& x_in_host,
                         const std::vector<float>& weight_host,
                         std::vector<float>& out_host, int n, float eps) {
  out_host.resize(n);
  float *x_dev = nullptr, *weight_dev = nullptr, *out_dev = nullptr;
  gpuErrchk(hipMalloc(&x_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&weight_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&out_dev, n * sizeof(float)));
  gpuErrchk(hipMemcpy(x_dev, x_in_host.data(), n * sizeof(float),
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(weight_dev, weight_host.data(), n * sizeof(float),
                       hipMemcpyHostToDevice));
  rmsnorm_vector_cuda(x_dev, weight_dev, out_dev, n, eps, 0);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(out_host.data(), out_dev, n * sizeof(float),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(x_dev));
  gpuErrchk(hipFree(weight_dev));
  gpuErrchk(hipFree(out_dev));
}

/**
 * @brief Implementation of matrix-vector multiplication with FP32
 * 
 * This function performs matrix-vector multiplication using cuBLAS,
 * optimized for FP32 precision. It handles the cuBLAS setup and
 * error checking.
 * 
 * @param handle cuBLAS handle
 * @param mat_f32_dev Matrix in row-major format (device pointer)
 * @param vec_f32_dev Vector to multiply (device pointer)
 * @param out_f32_dev Output vector (device pointer)
 * @param rows Number of matrix rows
 * @param cols Number of matrix columns
 * @param stream CUDA stream for asynchronous execution
 */
void matvec_f32_f32_cuda(hipblasHandle_t handle, const float* mat_f32_dev,
                         const float* vec_f32_dev, float* out_f32_dev, int rows,
                         int cols, hipStream_t stream) {
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int M = rows;
  int N = 1;
  int K = cols;

  hipblasStatus_t status = hipblasSetStream(handle, stream);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    Logger::error("hipblasSetStream failed");
    throw std::runtime_error("hipblasSetStream failed");
  }

  status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha,
                       mat_f32_dev, K, vec_f32_dev, K, &beta, out_f32_dev, M);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    Logger::error("hipblasSgemm (FP32) failed with status: " +
                  std::to_string(status));

    throw std::runtime_error("hipblasSgemm (FP32) failed");
  }
}

/**
 * @brief Implementation of matrix-vector multiplication with BF16
 * 
 * This function performs matrix-vector multiplication using cuBLAS,
 * with automatic conversion from BF16 to FP32. It's optimized for
 * models using Brain Floating Point format.
 * 
 * @param handle cuBLAS handle
 * @param mat_bf16_dev Matrix in BF16 format (device pointer)
 * @param vec_f32_dev Vector in FP32 format (device pointer)
 * @param out_f32_dev Output vector in FP32 format (device pointer)
 * @param rows Number of matrix rows
 * @param cols Number of matrix columns
 * @param stream CUDA stream for asynchronous execution
 */
void matvec_bf16_f32_cuda(hipblasHandle_t handle, const uint16_t* mat_bf16_dev,
                          const float* vec_f32_dev, float* out_f32_dev,
                          int rows, int cols, hipStream_t stream) {
  float* mat_fp32_dev = nullptr;
  size_t mat_size = (size_t)rows * cols;

  gpuErrchk(hipMalloc(&mat_fp32_dev, mat_size * sizeof(float)));

  const int threads_per_block_convert = 256;
  const int num_blocks_convert = (mat_size + threads_per_block_convert - 1) / threads_per_block_convert;
  convert_bf16_to_fp32_kernel<<<num_blocks_convert, threads_per_block_convert, 0, stream>>>(mat_bf16_dev, mat_fp32_dev, mat_size);
  gpuErrchk(hipGetLastError()); // Check for errors after kernel launch

  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasStatus_t status = hipblasSetStream(handle, stream);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    Logger::error("hipblasSetStream failed in matvec_bf16_f32_cuda fallback");
    gpuErrchk(hipFree(mat_fp32_dev));
    throw std::runtime_error("hipblasSetStream failed");
  }

  status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, 1, cols, &alpha,
                       mat_fp32_dev, cols, vec_f32_dev, cols, &beta,
                       out_f32_dev, rows);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    Logger::error("hipblasSgemm (BF16 fallback) failed with status: " +
                  std::to_string(status));
    gpuErrchk(hipFree(mat_fp32_dev));
    throw std::runtime_error("hipblasSgemm (BF16 fallback) failed");
  }

  gpuErrchk(hipFree(mat_fp32_dev));
}

void matvec_f32_f32_cuda(hipblasHandle_t handle,
                         const std::vector<float>& mat_f32_host,
                         const std::vector<float>& vec_f32_host,
                         std::vector<float>& out_f32_host, int rows, int cols) {
  if (mat_f32_host.size() != (size_t)rows * cols) {
    throw std::runtime_error(
        "matvec_f32_f32_cuda (host/host/host): mat size mismatch.");
  }
  if (vec_f32_host.size() != (size_t)cols) {
    throw std::runtime_error(
        "matvec_f32_f32_cuda (host/host/host): vec size mismatch.");
  }
  out_f32_host.resize(rows);

  float* mat_f32_dev = nullptr;
  float* vec_f32_dev = nullptr;
  float* out_f32_dev = nullptr;

  gpuErrchk(hipMalloc(&mat_f32_dev, mat_f32_host.size() * sizeof(float)));
  gpuErrchk(hipMalloc(&vec_f32_dev, vec_f32_host.size() * sizeof(float)));
  gpuErrchk(hipMalloc(&out_f32_dev, out_f32_host.size() * sizeof(float)));

  gpuErrchk(hipMemcpy(mat_f32_dev, mat_f32_host.data(),
                       mat_f32_host.size() * sizeof(float),
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vec_f32_dev, vec_f32_host.data(),
                       vec_f32_host.size() * sizeof(float),
                       hipMemcpyHostToDevice));

  matvec_f32_f32_cuda(handle, mat_f32_dev, vec_f32_dev, out_f32_dev, rows, cols,
                      0);

  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(out_f32_host.data(), out_f32_dev,
                       out_f32_host.size() * sizeof(float),
                       hipMemcpyDeviceToHost));

  gpuErrchk(hipFree(mat_f32_dev));
  gpuErrchk(hipFree(vec_f32_dev));
  gpuErrchk(hipFree(out_f32_dev));
}

__global__ void silu_kernel(const float* x, float* out, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float x_val = x[i];
    out[i] = x_val / (1.0f + expf(-x_val));
  }
}

void silu_cuda(const std::vector<float>& x_host, std::vector<float>& out_host,
               int n) {
  if (x_host.size() != n) {
    throw std::runtime_error("SiLU CUDA: Input vector size mismatch.");
  }
  out_host.resize(n);

  float* x_dev = nullptr;
  float* out_dev = nullptr;
  gpuErrchk(hipMalloc(&x_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&out_dev, n * sizeof(float)));

  gpuErrchk(hipMemcpy(x_dev, x_host.data(), n * sizeof(float),
                       hipMemcpyHostToDevice));

  const int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  silu_kernel<<<num_blocks, threads_per_block>>>(x_dev, out_dev, n);
  gpuErrchk(hipGetLastError());

  gpuErrchk(hipMemcpy(out_host.data(), out_dev, n * sizeof(float),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipFree(x_dev));
  gpuErrchk(hipFree(out_dev));
}

__global__ void softmax_find_max_kernel(const float* x, float* partial_max,
                                        int n) {
  extern __shared__ float sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? x[i] : -INFINITY;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();
  }

  if (tid == 0) {
    partial_max[blockIdx.x] = sdata[0];
  }
}

__global__ void softmax_exp_sum_kernel(const float* x, float* partial_sums,
                                       int n, float max_val) {
  extern __shared__ float sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? expf(x[i] - max_val) : 0.0f;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    partial_sums[blockIdx.x] = sdata[0];
  }
}

__global__ void softmax_normalize_kernel(const float* x, float* out, int n,
                                         float max_val, float inv_sum) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    out[i] = expf(x[i] - max_val) * inv_sum;
  }
}

void softmax_vector_cuda(const std::vector<float>& x_host,
                         std::vector<float>& out_host, int n) {
  if (x_host.size() != n) {
    throw std::runtime_error("Softmax CUDA: Input vector size mismatch.");
  }
  if (n == 0) {
    out_host.clear();
    return;
  }
  out_host.resize(n);

  float* x_dev = nullptr;
  float* out_dev = nullptr;
  float* partial_max_dev = nullptr;
  float* partial_sum_dev = nullptr;

  const int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  size_t shared_mem_size = threads_per_block * sizeof(float);

  float* partial_max_host = new float[num_blocks];
  float* partial_sum_host = new float[num_blocks];

  gpuErrchk(hipMalloc(&x_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&out_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&partial_max_dev, num_blocks * sizeof(float)));
  gpuErrchk(hipMalloc(&partial_sum_dev, num_blocks * sizeof(float)));

  gpuErrchk(hipMemcpy(x_dev, x_host.data(), n * sizeof(float),
                       hipMemcpyHostToDevice));

  softmax_find_max_kernel<<<num_blocks, threads_per_block, shared_mem_size>>>(
      x_dev, partial_max_dev, n);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(partial_max_host, partial_max_dev,
                       num_blocks * sizeof(float), hipMemcpyDeviceToHost));

  float max_val = -INFINITY;
  for (int i = 0; i < num_blocks; ++i) {
    if (partial_max_host[i] > max_val) {
      max_val = partial_max_host[i];
    }
  }

  softmax_exp_sum_kernel<<<num_blocks, threads_per_block, shared_mem_size>>>(
      x_dev, partial_sum_dev, n, max_val);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(partial_sum_host, partial_sum_dev,
                       num_blocks * sizeof(float), hipMemcpyDeviceToHost));

  double exp_sum = 0.0;
  for (int i = 0; i < num_blocks; ++i) {
    exp_sum += partial_sum_host[i];
  }
  float inv_sum = 1.0f / static_cast<float>(exp_sum);

  softmax_normalize_kernel<<<num_blocks, threads_per_block>>>(x_dev, out_dev, n,
                                                              max_val, inv_sum);
  gpuErrchk(hipGetLastError());

  gpuErrchk(hipMemcpy(out_host.data(), out_dev, n * sizeof(float),
                       hipMemcpyDeviceToHost));

  delete[] partial_max_host;
  delete[] partial_sum_host;
  gpuErrchk(hipFree(x_dev));
  gpuErrchk(hipFree(out_dev));
  gpuErrchk(hipFree(partial_max_dev));
  gpuErrchk(hipFree(partial_sum_dev));
}

__global__ void swiglu_kernel(const float* gate, const float* up, float* out,
                              int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float gate_val = gate[i];
    float silu_gate = gate_val / (1.0f + expf(-gate_val));
    out[i] = silu_gate * up[i];
  }
}

void swiglu_cuda(const std::vector<float>& gate_host,
                 const std::vector<float>& up_host,
                 std::vector<float>& out_host, int n) {
  if (gate_host.size() != n || up_host.size() != n) {
    throw std::runtime_error("SwiGLU CUDA: Input vector size mismatch.");
  }
  out_host.resize(n);

  float* gate_dev = nullptr;
  float* up_dev = nullptr;
  float* out_dev = nullptr;
  gpuErrchk(hipMalloc(&gate_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&up_dev, n * sizeof(float)));
  gpuErrchk(hipMalloc(&out_dev, n * sizeof(float)));

  gpuErrchk(hipMemcpy(gate_dev, gate_host.data(), n * sizeof(float),
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(up_dev, up_host.data(), n * sizeof(float),
                       hipMemcpyHostToDevice));

  const int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  swiglu_kernel<<<num_blocks, threads_per_block>>>(gate_dev, up_dev, out_dev,
                                                   n);
  gpuErrchk(hipGetLastError());

  gpuErrchk(hipMemcpy(out_host.data(), out_dev, n * sizeof(float),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipFree(gate_dev));
  gpuErrchk(hipFree(up_dev));
  gpuErrchk(hipFree(out_dev));
}

void swiglu_cuda(const float* gate_dev, const float* up_dev, float* out_dev,
                 int n, hipStream_t stream) {
  const int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  swiglu_kernel<<<num_blocks, threads_per_block, 0, stream>>>(gate_dev, up_dev,
                                                              out_dev, n);
  gpuErrchk(hipGetLastError());
}

__global__ void rope_kernel(float* x, int num_heads, int head_dim,
                            const float* all_freqs_cis_base, int pos, bool use_adjacent_pairing) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index for pairs
  int total_pairs = num_heads * (head_dim / 2);   // Total (dim_i, dim_{i+1}) or (dim_i, dim_{i+D/2}) pairs across all heads
  if (idx >= total_pairs) return;

  int head_idx = idx / (head_dim / 2);        // The specific head this thread works on
  int dim_pair_idx = idx % (head_dim / 2);  // The index of the pair within this head (0 to head_dim/2 - 1)

  size_t freq_cos_sin_pair_idx = (size_t)pos * (head_dim / 2) + dim_pair_idx;
  size_t freq_base_offset = freq_cos_sin_pair_idx * 2; 

  int base_x0, base_x1;
  if (use_adjacent_pairing) {
    // Adjacent pairing: Rotate x[h*HD + 2*j] with x[h*HD + 2*j + 1]
    base_x0 = head_idx * head_dim + (dim_pair_idx * 2);
    base_x1 = head_idx * head_dim + (dim_pair_idx * 2 + 1);
  } else {
    // Split-half pairing: Rotate x[h*HD + j] with x[h*HD + j + head_dim/2]
    base_x0 = head_idx * head_dim + dim_pair_idx;
    base_x1 = head_idx * head_dim + dim_pair_idx + (head_dim / 2);
  }

  float x0 = x[base_x0];
  float x1 = x[base_x1];
  
  float cos_val = all_freqs_cis_base[freq_base_offset];
  float sin_val = all_freqs_cis_base[freq_base_offset + 1];

  x[base_x0] = x0 * cos_val - x1 * sin_val;
  x[base_x1] = x0 * sin_val + x1 * cos_val;
}

void rope_cuda(float* x_dev, int num_heads, int head_dim,
               const float* all_freqs_cis_dev_base, int pos, bool use_adjacent_pairing, hipStream_t stream) {
  int total_pairs = num_heads * (head_dim / 2);
  int threads_per_block = 256;
  int num_blocks = (total_pairs + threads_per_block - 1) / threads_per_block;

  rope_kernel<<<num_blocks, threads_per_block, 0, stream>>>(
      x_dev, num_heads, head_dim, all_freqs_cis_dev_base, pos, use_adjacent_pairing);
  gpuErrchk(hipGetLastError());
}

__global__ void attention_kernel(const float* Q_current,
                                 const float* K_layer_cache_base,
                                 const float* V_layer_cache_base, float* out,
                                 int current_seq_len, int head_dim, float scale,
                                 int cache_num_kv_heads, int num_q_heads) {

  // Kernel launch: grid(num_q_heads), block(head_dim)
  // blockIdx.x is q_head_idx
  // threadIdx.x is d_idx (dimension index within head_dim)

  int q_head_idx = blockIdx.x;
  int d_idx = threadIdx.x; // dimension index for this thread

  if (q_head_idx >= num_q_heads) return; // Should not happen with correct launch

  // Determine the corresponding KV head for this Q head (for GQA/MQA)
  int heads_per_kv = num_q_heads / cache_num_kv_heads;
  int kv_head_idx = q_head_idx / heads_per_kv;

  // Shared memory for scores (one score per k_pos) and for dot product reduction
  extern __shared__ float shared_data[];
  float* scores = shared_data; // First part of shared_data: size current_seq_len
  float* dot_product_terms = &shared_data[current_seq_len]; // Second part: size head_dim (blockDim.x)

  // Pointer to the current query head's data
  const float* q_head_ptr = Q_current + q_head_idx * head_dim;

  // --- 1. Calculate Scores (Q_head dot K_k) ---
  // Each block (q_head) calculates all 'current_seq_len' scores.
  // The 'head_dim' threads in the block cooperate to calculate each dot product.
  for (int k_pos = 0; k_pos < current_seq_len; ++k_pos) {
    // Pointer to the k_pos-th key vector for the relevant kv_head
    size_t k_vec_offset = (size_t)k_pos * cache_num_kv_heads * head_dim +
                          (size_t)kv_head_idx * head_dim;
    const float* k_vec_ptr = K_layer_cache_base + k_vec_offset;

    // Parallel dot product calculation
    // Each thread d_idx computes one term of the dot product
    dot_product_terms[d_idx] = q_head_ptr[d_idx] * k_vec_ptr[d_idx];
    __syncthreads(); // Ensure all terms are written before reduction

    // Reduction in shared memory (e.g., by thread 0 or tree-based)
    if (d_idx == 0) {
      float current_dot = 0.0f;
      for (int r = 0; r < head_dim; ++r) {
        current_dot += dot_product_terms[r];
      }
      scores[k_pos] = current_dot * scale;
    }
    __syncthreads(); // Ensure scores[k_pos] is written before next k_pos iteration (if reduction by d_idx==0)
                     // or before softmax if reduction was parallel.
                     // If d_idx == 0 writes, all other threads must wait for it to complete for this k_pos.
  }

  // --- 2. Softmax ---
  // Parallel reduction to find max_score among scores[0...current_seq_len-1]
  // Each thread handles a portion of 'scores' array for reduction.
  float thread_max_score = -INFINITY;
  for (int i = d_idx; i < current_seq_len; i += blockDim.x) { // blockDim.x is head_dim
    if (scores[i] > thread_max_score) {
      thread_max_score = scores[i];
    }
  }
  // Reduce thread_max_score across the block
  // Simple single-pass reduction (can be optimized with tree reduction if head_dim is large)
  dot_product_terms[d_idx] = thread_max_score; // Repurpose dot_product_terms shared memory
  __syncthreads();

  float block_max_score = -INFINITY;
  if (d_idx == 0) {
    for (int r = 0; r < head_dim; ++r) { // head_dim might be > current_seq_len for first few tokens
      if (dot_product_terms[r] > block_max_score) { // Check if dot_product_terms[r] was validly written
        block_max_score = dot_product_terms[r];
      }
    }
  }
  __syncthreads();
  if(d_idx == 0) dot_product_terms[0] = block_max_score; // Store it.
  __syncthreads();
  block_max_score = dot_product_terms[0]; // All threads read it.


  // Calculate exp scores and sum
  float thread_exp_sum = 0.0f;
  for (int i = d_idx; i < current_seq_len; i += blockDim.x) {
    float prob = expf(scores[i] - block_max_score);
    scores[i] = prob; // Update scores in-place with exp(val - max)
    thread_exp_sum += prob;
  }
  // Reduce thread_exp_sum across the block
  dot_product_terms[d_idx] = thread_exp_sum; // Repurpose again
  __syncthreads();

  float block_exp_sum = 0.0f;
  if (d_idx == 0) {
    for (int r = 0; r < head_dim; ++r) { // Similar concern as max_score reduction range
      block_exp_sum += dot_product_terms[r];
    }
  }
  __syncthreads();
  // Store and reload block_exp_sum for all threads
  if(d_idx == 0) dot_product_terms[0] = block_exp_sum;
  __syncthreads();
  block_exp_sum = dot_product_terms[0];


  float inv_sum = 1.0f / (block_exp_sum + 1e-9f); // Add epsilon for stability

  // Normalize scores in shared memory
  for (int i = d_idx; i < current_seq_len; i += blockDim.x) {
    scores[i] *= inv_sum;
  }
  __syncthreads(); // Ensure all scores are normalized before weighted sum

  // --- 3. Weighted Sum of Values ---
  // Each thread d_idx calculates one element of the output vector for this q_head.
  // out_for_this_q_head[d_idx] = sum over k_pos ( scores[k_pos] * V_value_for_d_idx_at_k_pos )

  double weighted_val_d = 0.0; // Use double for accumulation
  for (int k_pos = 0; k_pos < current_seq_len; ++k_pos) {
    size_t v_vec_offset = (size_t)k_pos * cache_num_kv_heads * head_dim +
                          (size_t)kv_head_idx * head_dim;
    const float* v_vec_ptr = V_layer_cache_base + v_vec_offset;
    
    weighted_val_d += static_cast<double>(scores[k_pos]) * static_cast<double>(v_vec_ptr[d_idx]);
  }
  
  // Write the final result for this thread's dimension d_idx
  out[q_head_idx * head_dim + d_idx] = static_cast<float>(weighted_val_d);
}

void attention_cuda(const float* Q_current_dev, const float* K_layer_cache_base,
                    const float* V_layer_cache_base, float* out_dev,
                    int num_q_heads, int current_seq_len, int head_dim,
                    float scale, int cache_max_seq_len, int cache_num_kv_heads,
                    hipStream_t stream) {
  dim3 grid(num_q_heads);  // One block per Q head
  dim3 block(head_dim);   // head_dim threads per block

  // Shared memory: scores array (size current_seq_len) + dot_product_terms (size head_dim for reduction)
  size_t shared_mem_bytes = (current_seq_len + head_dim) * sizeof(float);
  

  attention_kernel<<<grid, block, shared_mem_bytes, stream>>>(
      Q_current_dev, K_layer_cache_base, V_layer_cache_base, out_dev,
      current_seq_len, head_dim, scale, cache_num_kv_heads, num_q_heads);
  gpuErrchk(hipGetLastError());
}

__global__ void add_vectors_kernel(const float* a, const float* b,
                                   float* result, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    result[i] = a[i] + b[i];
  }
}

void add_vectors_cuda(const float* a_dev, const float* b_dev, float* result_dev,
                      int n, hipStream_t stream) {
  const int threads_per_block = 256;
  const int num_blocks = (n + threads_per_block - 1) / threads_per_block;

  add_vectors_kernel<<<num_blocks, threads_per_block, 0, stream>>>(
      a_dev, b_dev, result_dev, n);
  gpuErrchk(hipGetLastError());
}

__global__ void add_residual_kernel(const float* matvec_out,
                                    const float* residual, float* result,
                                    int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    result[i] = matvec_out[i] + residual[i];
  }
}

void add_residual_cuda(const float* matvec_out_dev, const float* residual_dev,
                       float* result_dev, int n, hipStream_t stream) {
  const int threads_per_block = 256;
  const int num_blocks = (n + threads_per_block - 1) / threads_per_block;

  add_residual_kernel<<<num_blocks, threads_per_block, 0, stream>>>(
      matvec_out_dev, residual_dev, result_dev, n);
  gpuErrchk(hipGetLastError());
}

__global__ void update_kv_cache_kernel(float* cache_base_ptr,
                                       const float* current_kv_vector, int pos,
                                       int kv_head_idx, int max_seq_len,
                                       int num_kv_heads, int head_dim) {
  int d = threadIdx.x;
  if (d >= head_dim) return;

  size_t cache_offset = (size_t)pos * num_kv_heads * head_dim +
                        (size_t)kv_head_idx * head_dim + d;

  size_t source_offset = d;

  size_t total_cache_size = (size_t)max_seq_len * num_kv_heads * head_dim;
  if (cache_offset >= total_cache_size) {
    return;
  }

  cache_base_ptr[cache_offset] = current_kv_vector[source_offset];
}

void update_kv_cache_cuda(float* cache_base_ptr,
                          const float* current_kv_head_vector,

                          int pos, int kv_head_idx, int max_seq_len,
                          int num_kv_heads, int head_dim, hipStream_t stream) {
  dim3 blockDim(head_dim);
  dim3 gridDim(1);

  if (pos < 0 || pos >= max_seq_len) {
    Logger::error("update_kv_cache_cuda: pos out of bounds (" +
                  std::to_string(pos) + " >= " + std::to_string(max_seq_len) +
                  ")");
    return;
  }

  if (kv_head_idx < 0 || kv_head_idx >= num_kv_heads) {
    Logger::error("update_kv_cache_cuda: kv_head_idx out of bounds (" +
                  std::to_string(kv_head_idx) +
                  " >= " + std::to_string(num_kv_heads) + ")");
    return;
  }
  if (!current_kv_head_vector) {
    Logger::error("update_kv_cache_cuda: Input K/V vector pointer is null.");
    return;
  }

  update_kv_cache_kernel<<<gridDim, blockDim, 0, stream>>>(
      cache_base_ptr, current_kv_head_vector, pos, kv_head_idx, max_seq_len,
      num_kv_heads, head_dim);
  gpuErrchk(hipGetLastError());
}

__global__ void rope_and_update_kv_cache_kernel(
    float* cache_base_ptr, const float* kv_vector_head,

    const float* all_freqs_cis_base, int pos, int kv_head_idx, int max_seq_len,
    int num_kv_heads, int head_dim) {
  int pair_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int dim_half = head_dim / 2;

  if (pair_idx >= dim_half) return;

  int idx0 = pair_idx;
  int idx1 = pair_idx + dim_half;
  float kv0 = kv_vector_head[idx0];
  float kv1 = kv_vector_head[idx1];

  size_t freq_base_offset = (size_t)pos * head_dim + (size_t)pair_idx * 2;
  float cos_val = all_freqs_cis_base[freq_base_offset];
  float sin_val = all_freqs_cis_base[freq_base_offset + 1];

  float kv0_rotated = kv0 * cos_val - kv1 * sin_val;
  float kv1_rotated = kv0 * sin_val + kv1 * cos_val;

  size_t cache_offset_0 = (size_t)pos * num_kv_heads * head_dim +
                          (size_t)kv_head_idx * head_dim + idx0;
  size_t cache_offset_1 = cache_offset_0 + dim_half;

  size_t total_cache_size = (size_t)max_seq_len * num_kv_heads * head_dim;
  if (cache_offset_0 >= total_cache_size ||
      cache_offset_1 >= total_cache_size) {
    return;
  }

  cache_base_ptr[cache_offset_0] = kv0_rotated;
  cache_base_ptr[cache_offset_1] = kv1_rotated;
}

void rope_and_update_kv_cache_cuda(float* cache_base_ptr,
                                   const float* kv_vector_head,
                                   const float* all_freqs_cis_base, int pos,
                                   int kv_head_idx, int max_seq_len,
                                   int num_kv_heads, int head_dim,
                                   hipStream_t stream) {
  if (head_dim % 2 != 0) {
    Logger::error("rope_and_update_kv_cache_cuda: head_dim must be even.");
    return;
  }
  if (pos < 0 || pos >= max_seq_len) {
    Logger::error("rope_and_update_kv_cache_cuda: pos out of bounds.");
    return;
  }
  if (kv_head_idx < 0 || kv_head_idx >= num_kv_heads) {
    Logger::error("rope_and_update_kv_cache_cuda: kv_head_idx out of bounds.");
    return;
  }
  if (!kv_vector_head || !all_freqs_cis_base || !cache_base_ptr) {
    Logger::error(
        "rope_and_update_kv_cache_cuda: Received null device pointer(s).");
    return;
  }

  int threads_per_block = 128;
  int num_blocks = (head_dim / 2 + threads_per_block - 1) / threads_per_block;

  rope_and_update_kv_cache_kernel<<<num_blocks, threads_per_block, 0, stream>>>(
      cache_base_ptr, kv_vector_head, all_freqs_cis_base, pos, kv_head_idx,
      max_seq_len, num_kv_heads, head_dim);
  gpuErrchk(hipGetLastError());
}

__global__ void lookup_embedding_kernel(const void* __restrict__ table_dev,
                                        float* __restrict__ output_dev,
                                        int token_id, int hidden_size,
                                        int vocab_size, bool is_bf16) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= hidden_size) {
    return;
  }

  if (token_id < 0 || token_id >= vocab_size) {
    output_dev[idx] = 0.0f;
    return;
  }
  size_t offset = (size_t)token_id * hidden_size + idx;

  if (is_bf16) {
    const uint16_t* table_bf16 = static_cast<const uint16_t*>(table_dev);

    uint16_t val_bf16 = table_bf16[offset];

    output_dev[idx] = bf16_to_float32_device(val_bf16);
  } else {
    const float* table_f32 = static_cast<const float*>(table_dev);

    output_dev[idx] = table_f32[offset];
  }
}

void lookup_embedding_cuda(const void* table_dev, float* output_dev,
                           int token_id, int hidden_size, int vocab_size,
                           bool is_bf16, hipStream_t stream) {
  if (!table_dev || !output_dev) {
    Logger::error("lookup_embedding_cuda: Received null device pointer(s).");

    return;
  }
  if (hidden_size <= 0 || vocab_size <= 0) {
    Logger::error("lookup_embedding_cuda: Invalid hidden_size or vocab_size.");
    return;
  }

  int threads_per_block = 256;

  int blocks_per_grid =
      (hidden_size + threads_per_block - 1) / threads_per_block;

  lookup_embedding_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(
      table_dev, output_dev, token_id, hidden_size, vocab_size, is_bf16);

  gpuErrchk(hipGetLastError());
}

__global__ void convert_bf16_to_fp32_kernel(const uint16_t* __restrict__ bf16_in,
                                            float* __restrict__ fp32_out,
                                            size_t n_elements) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_elements) {
    fp32_out[idx] = bf16_to_float32_device(bf16_in[idx]);
  }
}

#endif